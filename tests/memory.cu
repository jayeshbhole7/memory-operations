#include <iostream>
#include <hip/hip_runtime.h>
#include <cassert>

__global__ void test_kernal(int* input,int* output,int arr_io){

}

void test_memory_read_write(){
    const int s_of_memory_allotted = 1024;

    int h_input[s_of_memory_allotted], h_output[s_of_memory_allotted];

    for(int i=0; i<s_of_memory_allotted;i++) h_input[i]=i;

    int* d_input, *d_output;
    hipMalloc(&d_input,s_of_memory_allotted*sizeof(int));
    hipMalloc(&d_output,s_of_memory_allotted*sizeof(int));

    hipMemcpy(d_input, h_input, s_of_memory_allotted * sizeof(int), hipMemcpyHostToDevice);

    test_kernal<<<s_of_memory_allotted/256, 256>>>(d_input,d_output,s_of_memory_allotted);

    hipMemcpy(h_output, h_input, s_of_memory_allotted* sizeof(int),hipMemcpyDeviceToHost);

    for(int i=0;i<s_of_memory_allotted; i++) assert(h_output[i]== h_input[i]*2);

    hipFree(d_input);
    hipFree(d_output);
}

int main(){
    test_memory_read_write();
}