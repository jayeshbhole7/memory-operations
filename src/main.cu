#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

int main{

    const int n =1024;

    int a[n],b[n];

    for(int i=0;i<n;i++) a[i]=i;

    int *d_a,*d_b;

    hipMalloc(&d_a, n*sizeof(int));
    hipMalloc(&d_b, n*sizeof(int));

    hipMemcpy(d_a,a,n*sizeof(int), hipMemcpyDeviceToHost);
    
    memory_read_write<<n/256,256>>(d_a,d_b,n);

    hipMemcpy(b,d_b,n*sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0;i<10;i++) cout<<b[i]<<"";

    hipFree(d_a);
    hipFree(d_b);

    return 0;
}
