#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;


__global__ void memory_read_write(int *a,int *b,int n){
}

int main(){

    const int n =1024;

    int a[n],b[n];

    for(int i=0;i<n;i++) a[i]=i;

    int *d_a,*d_b;

    if(hipMalloc(&d_a,n*sizeof(int))!=hipSuccess){
        cerr<<"cuda malloc fails for d_a\n";
        return -1;
    }
    if(hipMalloc(&d_a,n*sizeof(int))!=hipSuccess){
        cerr<<"cuda malloc fails for d_b\n";
        return -1;
    }

    hipMemcpy(d_a,a,n*sizeof(int), hipMemcpyHostToDevice);
    
    memory_read_write<<<n/256,256>>>(d_a,d_b,n);

    hipMemcpy(b,d_b,n*sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0;i<n;i++) cout<<b[i]<<" ";

    hipFree(d_a);
    hipFree(d_b);

    return 0;
}
