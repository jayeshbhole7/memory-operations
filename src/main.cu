#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;


__global__ void memory_read_write(int* input,int* output,int n){
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    if(index < n){
        output[index] = input[index]*2; //read_write_memory
    }
}

int main(){

    const int s_of_data_processed =1024;

    int h_input[s_of_data_processed],h_output[s_of_data_processed];

    for(int i=0;i<s_of_data_processed;i++) h_input[i]=i;

    int* d_input,*d_output;
    hipMalloc(&d_input,s_of_data_processed*sizeof(int));
    hipMalloc(&d_output,s_of_data_processed*sizeof(int));


    hipMemcpy(d_input,h_input,s_of_data_processed*sizeof(int), hipMemcpyHostToDevice);
    
    memory_read_write<<<s_of_data_processed/256,256>>>(d_input,d_output,s_of_data_processed);

    hipMemcpy(h_output,d_output,s_of_data_processed*sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0;i<s_of_data_processed;i++) cout<<h_output[i]<<" ";

    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
