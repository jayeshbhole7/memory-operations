#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;


__global__ void memory_read_write(int* input,int* output,int n){
}

int main(){

    const int s_of_data_processed =1024;

    int input[s_of_data_processed],output[s_of_data_processed];

    for(int i=0;i<s_of_data_processed;i++) input[i]=i;

    int *d_input,*d_output;

    if(hipMalloc(&d_input,s_of_data_processed*sizeof(int))!=hipSuccess){
        cerr<<"cuda malloc fails for d_a\n";
        return -1;
    }
    if(hipMalloc(&d_output,s_of_data_processed*sizeof(int))!=hipSuccess){
        cerr<<"cuda malloc fails for d_b\n";
        return -1;
    }

    hipMemcpy(d_input,input,s_of_data_processed*sizeof(int), hipMemcpyHostToDevice);
    
    memory_read_write<<<s_of_data_processed/256,256>>>(d_input,d_output,s_of_data_processed);

    hipMemcpy(output,d_output,s_of_data_processed*sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0;i<s_of_data_processed;i++) cout<<output[i]<<" ";

    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
